#include "hip/hip_runtime.h"
#include "log_helpers.cuh"

void __global__ driver() {
  float fast_res = fast_log2(3.14f);
  float precise_res = precise_log2(3.14f);
}

int main() {
  driver<<<1, 1>>>();
}